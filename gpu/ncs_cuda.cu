#include "hip/hip_runtime.h"
/*
 * CUDA kernel code for NCS.
 *
 * Hazen 08/19
 */

/*
 * Run NCS noise reduction on sub-regions.
 * 
 * Note: Any zero or negative values in the sub-regions should be
 *       set to a small positive value like 1.0.
 *
 * data_in - Sub-region data in e-.
 * g_gamma - Sub-region CMOS variance in units of e-^2.
 * otf_mask - 16 x 16 array containing the OTF mask.
 * data_out - Storage for noise corrected sub-regions.
 * iterations - Number of L-BFGS solver iterations.
 * status - Status of the solution (good, failed because of X).
 * alpha - NCS alpha term.
 */
__global__ void ncsReduceNoise(float4 *data_in,
                               float4 *g_gamma,
			       float4 *otf_mask,
			       float4 *data_out,
			       int *g_iterations,
			       int *g_status,
			       float alpha)
{
    int g_id = threadIdx.x;
    int offset = g_id*PSIZE;

    /* Variables. */
    int i;
    int iterations;
    int status;
    
    float4 data[PSIZE];
    float4 gamma[PSIZE];
    float4 otf_mask_sqr[PSIZE];
    float4 u_r[PSIZE]; 

    /* Initialization. */    
    for (i=0; i<PSIZE; i++){
        data[i] = data_in[i + offset];
        gamma[i] = g_gamma[i + offset];
        otf_mask_sqr[i] = otf_mask[i] * otf_mask[i];
        u_r[i] = data_in[i + offset];
    }

    /* Run NCS calculation. */
    ncsReduceNoiseSR(data, gamma, otf_mask_sqr, u_r, &iterations, &status, alpha);

    /* Save results. */
    for (i=0; i<PSIZE; i++){
    	data_out[i + offset] = u_r[i];
    }
    g_iterations[g_id] = iterations;
    g_status[g_id] = status;
}
